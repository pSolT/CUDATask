#include "hip/hip_runtime.h"
#include "helper_functions.cuh"

#define BLOCK_SIZE 1024

// Use dynamic shared memory
__global__ void solverKernel(float* A, float* x, float* result, size_t n)
{
    // Use dynamically allocated shared memory
    int tId = blockIdx.y * gridDim.x +  blockIdx.x * blockDim.x + threadIdx.x; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float cValue = A[tId] * x[col];
    atomicAdd(&result[blockIdx.y], cValue);
}

float SolveSODE_CUDA(float* A, float* x, float* result, size_t N)
{
    float *dA, *dx, *dResult;
    // Allocate the memory and memset result to 0
    checkCudaErrors(hipMalloc(&dA, N*N*sizeof(float)));
    checkCudaErrors(hipMemcpy(dA, A, N*N*sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&dx, N*sizeof(float)));
    checkCudaErrors(hipMemcpy(dx, x, N*sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&dResult, N*sizeof(float)));
    checkCudaErrors(hipMemset(dResult, 0, N*sizeof(float)));

    dim3 blockSize(BLOCK_SIZE, 1, 1);
    dim3 gridSize(N/BLOCK_SIZE, N, 1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, NULL));
    solverKernel<<<blockSize, gridSize>>>(dA, dx, dResult, N);  
    checkCudaErrors(hipEventRecord(stop, NULL));
    
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTimeMs = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeMs, start, stop));

    checkCudaErrors(hipMemcpy(result, dResult, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dx));
    checkCudaErrors(hipFree(dResult));
    
    return elapsedTimeMs;
}

float SolveSODE_CUBLAS(float* A, float* x, float* result, size_t N)
{
    float *dA, *dx, *dResult;
    // Allocate the memory and memset result to 0
    checkCudaErrors(hipMalloc(&dA, N*N*sizeof(float)));
    checkCudaErrors(hipMalloc(&dx, N*sizeof(float)));
    checkCudaErrors(hipMalloc(&dResult, N*sizeof(float)));
    checkCudaErrors(hipMemset(dResult, 0, N*sizeof(float)));

    cublasCheckErrors(hipblasSetVector(N, sizeof(float), x, 1, dx, 1));
    cublasCheckErrors(hipblasSetMatrix(N, N, sizeof(float), A, N, dA, N));

    hipblasHandle_t handle;
    cublasCheckErrors(hipblasCreate(&handle));
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    float alpha = (-1.0f); // We want a negative
    float beta = 0.0f;

    checkCudaErrors(hipEventRecord(start, NULL));
    cublasCheckErrors(hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, dA, N, dx, 1, &beta, dResult, 1));
    checkCudaErrors(hipEventRecord(stop, NULL));
    
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTimeMs = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeMs, start, stop));
    
    cublasCheckErrors(hipblasGetVector(N, sizeof(float), dResult, 1, result, 1));

    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dx));
    checkCudaErrors(hipFree(dResult));
    
    return elapsedTimeMs;
}