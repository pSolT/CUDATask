#include "hip/hip_runtime.h"
#include "helper_functions.cuh"

#define BLOCK_SIZE 1024


__global__ void solverKernel(float* A, float* x, float* result, size_t n, unsigned t, float h)
{
    __shared__ float ATile[BLOCK_SIZE];
    __shared__ float xTile[BLOCK_SIZE];
    float cValue = 1.0f;

    int aIdx = n * blockIdx.y + blockIdx.x * blockDim.x + threadIdx.x;
    int xIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Be careful not to go out of bounds
    if(xIdx < n)
    {
        // Copy row of A and x into shared memory in order to minimize the reads
        ATile[threadIdx.x] = A[aIdx];
        xTile[threadIdx.x] = x[xIdx];
        __syncthreads();

        for(int i=0; i<t; i++)
        {
            cValue += h * (ATile[aIdx] * xTile[xIdx]);
        }
        atomicAdd(&result[blockIdx.y], cValue);
    }
}

float SolveSODE_CUDA(float* A, float* x, float* result, size_t N, unsigned t)
{
    float *dA, *dx, *dResult;
    // Allocate the memory and memset result to 0
    checkCudaErrors(hipMalloc(&dA, N*N*sizeof(float)));
    checkCudaErrors(hipMemcpy(dA, A, N*N*sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&dx, N*sizeof(float)));
    checkCudaErrors(hipMemcpy(dx, x, N*sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&dResult, N*sizeof(float)));
    checkCudaErrors(hipMemset(dResult, 0, N*sizeof(float)));

    dim3 blockSize(BLOCK_SIZE, 1, 1);
    int numBlocksX = ceil(float(N)/BLOCK_SIZE);
    dim3 gridSize(numBlocksX, N, 1);

    printf("Grid : {%d, %d, %d} blocks. Blocks : {%d, %d, %d} threads.\n",
            gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    checkCudaErrors(hipEventRecord(start, NULL));


    float h = 0.01;
    solverKernel<<<blockSize, gridSize>>>(dA, dx, dResult, N, t, h);  

    checkCudaErrors( hipPeekAtLastError() );
    checkCudaErrors( hipDeviceSynchronize() );
    checkCudaErrors(hipEventRecord(stop, NULL));
    
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTimeMs = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeMs, start, stop));

    checkCudaErrors(hipMemcpy(result, dResult, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dx));
    checkCudaErrors(hipFree(dResult));
    
    return elapsedTimeMs;
}

float SolveSODE_CUBLAS(float* A, float* x, float* result, size_t N, unsigned t)
{
    float *dA, *dx, *dResult;
    // Allocate the memory and memset result to 0
    checkCudaErrors(hipMalloc(&dA, N*N*sizeof(float)));
    checkCudaErrors(hipMalloc(&dx, N*sizeof(float)));
    checkCudaErrors(hipMalloc(&dResult, N*sizeof(float)));
    checkCudaErrors(hipMemset(dResult, 0, N*sizeof(float)));

    cublasCheckErrors(hipblasSetVector(N, sizeof(float), x, 1, dx, 1));
    cublasCheckErrors(hipblasSetMatrix(N, N, sizeof(float), A, N, dA, N));

    hipblasHandle_t handle;
    cublasCheckErrors(hipblasCreate(&handle));
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    float alpha = (-1.0f); // We want a negative
    float beta = 0.0f;

    checkCudaErrors(hipEventRecord(start, NULL));
    cublasCheckErrors(hipblasSgemv(handle, HIPBLAS_OP_T, N, N, &alpha, dA, N, dx, 1, &beta, dResult, 1));
    checkCudaErrors(hipEventRecord(stop, NULL));
    
    checkCudaErrors(hipEventSynchronize(stop));
    float elapsedTimeMs = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeMs, start, stop));
    
    cublasCheckErrors(hipblasGetVector(N, sizeof(float), dResult, 1, result, 1));

    checkCudaErrors(hipFree(dA));
    checkCudaErrors(hipFree(dx));
    checkCudaErrors(hipFree(dResult));
    
    return elapsedTimeMs;
}